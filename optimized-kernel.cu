
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define TILE_SIZE  8
	__global__ void csr_multiply(int* d_row_size ,int* d_sparse, int*  d_cols, int* d_vals, int* d_X, int* d_y){
__shared__  int dot[TILE_SIZE];
int thid = TILE_SIZE * blockIdx.x + threadIdx.x;
int warp = thid/32;
int stride = thid % 32;
int row = warp;
		if (row < *d_row_size) {
		dot[threadIdx.x]=0;
		int row_start = d_sparse[row];
		int row_end = d_sparse[row+1];
			for(int elem = row_start+stride; elem < row_end ; elem +=32){
				dot[threadIdx.x]+= d_vals[elem] * d_X[d_cols[elem]];
	
}

for ( int s = 32 >> 1 ; s >= 1; s >>=1) {

if(stride < s ){dot[threadIdx.x] += dot[threadIdx.x +s];}

}

       if(stride==0){
                d_y[row]+=dot[threadIdx.x];

}



}
}
