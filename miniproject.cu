#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include "support.h"
#include <stdlib.h>

__global__ void check(int * d_out , int * d_in) {
	
	extern __shared__ int  ionCurrent[];
		int d_NUM_DUST=10;
        	int thid = threadIdx.x;
       		 while(thid <d_NUM_DUST){
               	 ionCurrent[thid]=0;
               	 thid+=blockDim.x;
        	}
        __syncthreads();
int idx = blockIdx.x*blockDim.x + threadIdx.x;
if( d_in[idx] > 0 )
{int  R = d_in[idx]-1 ;
atomicAdd(&(ionCurrent[R]),1);
d_out[R]=ionCurrent[R];
//d_out[R]+=1;
}

}

int main(int argc, char ** argv) {

 Timer timer;
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
startTime(&timer);

    // Initialize host variables ----------------------------------------------

   
int  h_NUM_DUST = 1000000;
int  ionCurrent[h_NUM_DUST];
	for (int k = 0; k <h_NUM_DUST; k++){
         ionCurrent[k] = 0;
          }
int boundsIon[h_NUM_DUST];
     for(int k=0; k<h_NUM_DUST; k++){
        if((rand()%5) <= 2){boundsIon[k]=0;}
            else {
        boundsIon[k]=rand()% h_NUM_DUST;}

       // const int ARRAY_BYTES =h_NUM_DUST *  sizeof(int);

        // declare GPU memory pointers
int * d_in;
int * d_out;

//stopTime(&timer); printf("%f s\n", elapsedTime(timer));

//allocate GPU memory;



cuda_ret = hipMalloc((void **) &d_in, ARRAY_BYTES);
if(cuda_ret !=hipSuccess) FATAL("unable to allocate device memory" );
cuda_ret =hipMalloc((void **) &d_out, ARRAY_BYTES);
if(cuda_ret !=hipSuccess) FATAL("unable to allocate device memory" );

 hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));


// transfer the array to the GPU
  printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

cuda_ret = hipMemcpy(d_in, boundsIon  , ARRAY_BYTES, hipMemcpyHostToDevice);
 if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

 hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));


//launch the kernel
 printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

const unsigned int THREADS_PER_BLOCK = 512;
const unsigned int numBlocks = (h_NUM_DUST-1)/THREADS_PER_BLOCK +1;
dim3 gridDim(numBlocks, 1, 1), blockDim(THREADS_PER_BLOCK, 1, 1);
check<<<numBlocks, THREADS_PER_BLOCK, ARRAY_BYTES>>>(d_out,d_in);

cuda_ret = hipDeviceSynchronize();
        if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

//copy back the result array to the cpu
  printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);
 

cuda_ret = hipMemcpy(ionCurrent , d_out , ARRAY_BYTES, hipMemcpyDeviceToHost);
if(cuda_ret != hipSuccess) FATAL("Unable to copy memory from device");

hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

printf("time of the end printing ...."); fflush(stdout);
    startTime(&timer);



hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));*/
//free GPU memory allocation
    printf("Verifying results..."); fflush(stdout);

    verify(A_h, B_h, C_h, matArow, matAcol, matBcol);*/

hipFree(d_in);
hipFree(d_out);
stopTime(&timer); printf("%f s\n", elapsedTime(timer));
return 0;
}

