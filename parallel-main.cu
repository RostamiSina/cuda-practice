#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"
#include <sys/time.h>
#include "support.h"
#define TILE_SIZE 8
int main(int argc, char ** argv ) 
{
	 Timer timer;
    hipError_t cuda_ret;
	  printf("\nSetting up the problem..."); fflush(stdout);
startTime(&timer);
int count=0;
     int length=70;
    int width=70;
    int Matrix[length][width];
    for(int i=0; i < length ; i++) {
     for (int j=0; j< width ; j++){
            if((rand()%5) <= 3){Matrix[i][j]=0;}
            else {
        Matrix[i][j]=rand()%60; if (Matrix[i][j] > 0){count++;}}
       // printf(" %d\t", Matrix[i][j]);
}
     }
     int X[length];
     for(int k=0; k<length; k++){
        if((rand()%5) >= 3){X[k]=0;}
            else {
        X[k]=rand()%60;}
        //printf(" \n X val  %d\t", X[k]);
}

    int rows[count];
int cols[count];
int vals[count];
    int sparse[count];
    int k=1;
    int f=1;
    int t=0;
    int row_size=sizeof(Matrix)/sizeof(Matrix[0]);
    int col_size=sizeof(Matrix[0])/sizeof(Matrix[0][0]);
    //printf("Row size of Matrix is %d \n", row_size);
    //printf("column size of Matrix is %d ", col_size);
    for (int i = 0; i < row_size; i++) {
                              for(int j=0; j <col_size ; j++){
                                        if (Matrix[i][j] != 0) {
            rows[t] = i;
            cols[t] = j;
            vals[t] = Matrix[i][j];
            //printf("\n %d \t %d \t %d", rows[t], cols[t], vals[t]);
                         t++;}
                                     }

                                        }
int   r_start, r_end;
 sparse[0]=0;
for (int row = 0; row < count-1 ; row++){
                    r_start=rows[row];
                    r_end=rows[row+1];
                    if (r_end > r_start){
sparse[k]=f;
    k++;} else {
		
			f++;
                    }
                                        }
sparse[k]=count;
                          int colsizer=sizeof(vals)/sizeof(vals[0]);
                                        //printf("\n your col size is %d",colsizer);
                                        sparse[k] = colsizer;
/*for( int l=0 ; l < count  ; l++) {
printf("\n  the sparse is %d  \n", sparse[l]);}*/
	int y[length];
	int * d_vals;
	int * d_cols;
	int * d_row_size;
	int * d_sparse;
	int * d_X;
	int * d_y;
	const int ARRAY_BYTES =colsizer *  sizeof(int);
	stopTime(&timer); printf("%f s\n", elapsedTime(timer));

//allocate GPU memory;

 	printf("Allocating device variables..."); fflush(stdout);
    	startTime(&timer);

	cuda_ret = hipMalloc((void **) &d_vals, ARRAY_BYTES);
	if(cuda_ret !=hipSuccess) FATAL("unable to allocate device memory" );
	cuda_ret =hipMalloc((void **) &d_cols, ARRAY_BYTES);
	if(cuda_ret !=hipSuccess) FATAL("unable to allocate device memory" );
	cuda_ret = hipMalloc((void **) &d_row_size, ARRAY_BYTES);
        if(cuda_ret !=hipSuccess) FATAL("unable to allocate device memory" );
        cuda_ret =hipMalloc((void **) &d_sparse, ARRAY_BYTES);
        if(cuda_ret !=hipSuccess) FATAL("unable to allocate device memory" );
 	cuda_ret = hipMalloc((void **) &d_y, ARRAY_BYTES);
        if(cuda_ret !=hipSuccess) FATAL("unable to allocate device memory" );
        cuda_ret =hipMalloc((void **) &d_X, ARRAY_BYTES);
        if(cuda_ret !=hipSuccess) FATAL("unable to allocate device memory" );
    	stopTime(&timer); printf("%f s\n", elapsedTime(timer));
	hipDeviceSynchronize();


	// transfer the array to the GPU
	  printf("Copying data from host to device..."); fflush(stdout);
   	 startTime(&timer);

	cuda_ret = hipMemcpy(d_vals, vals  , ARRAY_BYTES, hipMemcpyHostToDevice);
	 if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");
	cuda_ret = hipMemcpy(d_cols, cols  , ARRAY_BYTES, hipMemcpyHostToDevice);
         if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");
	cuda_ret = hipMemcpy(d_sparse, sparse  , ARRAY_BYTES, hipMemcpyHostToDevice);
         if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");
	cuda_ret = hipMemcpy(d_X, X  , ARRAY_BYTES, hipMemcpyHostToDevice);
         if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");
	cuda_ret = hipMemcpy(d_row_size, &row_size  , ARRAY_BYTES, hipMemcpyHostToDevice);
         if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");
 	hipDeviceSynchronize();
    	stopTime(&timer); printf("%f s\n", elapsedTime(timer));
// luanching kernel
	 printf("Launching kernel..."); fflush(stdout);
    	startTime(&timer);

	//const unsigned int THREADS_PER_BLOCK = 512;
	const unsigned int numBlocks = (row_size*32-1)/TILE_SIZE +1;
	dim3 gridDim(numBlocks, 1, 1), blockDim(TILE_SIZE, 1, 1);
	csr_multiply<<<numBlocks, TILE_SIZE, ARRAY_BYTES>>>(d_row_size, d_sparse, d_cols, d_vals, d_X, d_y);

	cuda_ret = hipDeviceSynchronize();
        if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    	stopTime(&timer); printf("%f s\n", elapsedTime(timer));

	printf("Copying data from device to host..."); fflush(stdout);
    	startTime(&timer);


	cuda_ret = hipMemcpy(y , d_y , ARRAY_BYTES, hipMemcpyDeviceToHost);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory from device");

	hipDeviceSynchronize();
    	stopTime(&timer); printf("%f s\n", elapsedTime(timer));

	//printf("time of the end printing ...."); fflush(stdout);
    	//startTime(&timer);
	
	hipFree(d_row_size);
        hipFree(d_cols);
	hipFree(d_sparse);
	hipFree(d_sparse);
	hipFree(d_y);
        hipFree(d_X);
/*for (int s=0 ; s<length ; s++){
printf(" \n your Y value is  %d ", y[s]);}*/ 
    return 0;
}
