
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

	__global__ void csr_multiply(int *d_row_size, int* d_sparse, int*  d_cols, int* d_vals, int* d_X, int* d_y){
	int row = blockDim.x*blockIdx.x+threadIdx.x;


		if (row < *d_row_size) {
		int dot = 0;
		int row_start = d_sparse[row];
		int row_end = d_sparse[row+1];
			for(int elem = row_start; elem < row_end ; elem ++){
				dot += (d_vals[elem]) *( d_X[d_cols[elem]]);

	}		
	d_y[row]+=dot;

}
}
