#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "support.h"
#define TILE_SIZE 8
int main(int argc, char ** argv ) 
{
	 Timer timer;
    hipError_t cuda_ret;
	  printf("\nSetting up the problem..."); fflush(stdout);
startTime(&timer);
  int length=70;
    int width=70;
    int Matrix[length][width];
    for(int i=0; i < length ; i++) {
     for (int j=0; j< width ; j++){
            if((rand()%5) <= 3){Matrix[i][j]=0;}
            else {
        Matrix[i][j]=rand()%60;}
        }
     }
     int X[length];
     for(int k=0; k<length; k++){
        if((rand()%5) >= 3){X[k]=0;}
            else {
        X[k]=rand()%60;}
        }


int J[length];
for (int o =0 ; o < length ; o++ ){
        J[o]=0;
    for(int u=0; u < width ; u ++){
        J[o]+=X[u]*Matrix[o][u];
    }
}

        
	stopTime(&timer); printf("%f s\n", elapsedTime(timer));

/*for (int s=0 ; s<length ; s++){
printf(" \n your Y value is  %d ", y[s]);}*/ 
    return 0;
}
