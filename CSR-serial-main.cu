#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "support.h"
#define TILE_SIZE 8
int main(int argc, char ** argv ) 
{
	 Timer timer;
    hipError_t cuda_ret;
	  printf("\nSetting up the problem..."); fflush(stdout);
startTime(&timer);
int count=0;
     int length=80;
    int width=80;
    int Matrix[length][width];
    for(int i=0; i < length ; i++) {
     for (int j=0; j< width ; j++){
            if((rand()%5) <= 3){Matrix[i][j]=0;}
            else {
        Matrix[i][j]=rand()%60; if (Matrix[i][j] > 0){count++;}}
       // printf(" %d\t", Matrix[i][j]);
}
     }
     int X[length];
     for(int k=0; k<length; k++){
        if((rand()%5) >= 3){X[k]=0;}
            else {
        X[k]=rand()%60;}
        //printf(" \n X val  %d\t", X[k]);
}

    int rows[count];
int cols[count];
int vals[count];
    int sparse[count];
    int k=1;
    int f=1;
    int t=0;
    int row_size=sizeof(Matrix)/sizeof(Matrix[0]);
    int col_size=sizeof(Matrix[0])/sizeof(Matrix[0][0]);
    //printf("Row size of Matrix is %d \n", row_size);
    //printf("column size of Matrix is %d ", col_size);
    for (int i = 0; i < row_size; i++) {
                              for(int j=0; j <col_size ; j++){
                                        if (Matrix[i][j] != 0) {
            rows[t] = i;
            cols[t] = j;
            vals[t] = Matrix[i][j];
            //printf("\n %d \t %d \t %d", rows[t], cols[t], vals[t]);
                         t++;}
                                     }

                                        }
int   r_start, r_end;
 sparse[0]=0;
for (int row = 0; row < count-1 ; row++){
                    r_start=rows[row];
                    r_end=rows[row+1];
                    if (r_end > r_start){
sparse[k]=f;
    k++;} else {
		
			f++;
                    }
                                        }
sparse[k]=count;
                          int colsizer=sizeof(vals)/sizeof(vals[0]);
                                        //printf("\n your col size is %d",colsizer);
                                        sparse[k] = colsizer;
/*for( int l=0 ; l < count  ; l++) {
printf("\n  the sparse is %d  \n", sparse[l]);}*/
int y[length];


for (int row = 0; row < length; row++){
                    r_start=sparse[row];
                    r_end=sparse[row+1];
                    int dot=0;
                    for(int elem=r_start; elem<r_end; elem++){
                          

                        dot += vals[elem]*X[cols[elem]];
                        //printf(" \n prime result is %d ", dot);

                    }
 y[row] += dot ;
 //printf(" \n your multiply is %d ", y[row]);
 }
	stopTime(&timer); printf("%f s\n", elapsedTime(timer));

/*for (int s=0 ; s<length ; s++){
printf(" \n your Y value is  %d ", y[s]);}*/ 
    return 0;
}
